#include <stdlib.h>
#include <stdio.h>
#include <assert.h>

#include <hip/hip_runtime.h>
#include <hiptensor.h>

#include <unordered_map>
#include <vector>
#include "contraction.cuh"

// Handle cuTENSOR errors
#define HANDLE_ERROR(x)                                         \
    {                                                           \
        const auto err = x;                                     \
        if (err != HIPTENSOR_STATUS_SUCCESS)                     \
        {                                                       \
            printf("Error: %s\n", hiptensorGetErrorString(err)); \
            exit(-1);                                           \
        }                                                       \
    };

#define HANDLE_CUDA_ERROR(x)                                \
    {                                                       \
        const auto err = x;                                 \
        if (err != hipSuccess)                             \
        {                                                   \
            printf("Error: %s\n", hipGetErrorString(err)); \
            exit(-1);                                       \
        }                                                   \
    };

void performContraction(std::vector<int> modeC, std::vector<int> modeA, std::vector<int> modeB, std::unordered_map<int, int64_t> extent)
{
    // Host element type definition
    typedef float floatTypeA;
    typedef float floatTypeB;
    typedef float floatTypeC;
    typedef float floatTypeCompute;

    // CUDA types
    hiptensorDataType_t typeA = HIPTENSOR_R_32F;
    hiptensorDataType_t typeB = HIPTENSOR_R_32F;
    hiptensorDataType_t typeC = HIPTENSOR_R_32F;
    hiptensorComputeDescriptor_t descCompute = HIPTENSOR_COMPUTE_DESC_32F;

    printf("Include headers and define data types\n");

    /* ***************************** */

    // Create vector of modes
    // std::vector<int> modeC{'m', 'u', 'n', 'v'};
    // std::vector<int> modeA{'m', 'h', 'k', 'n'};
    // std::vector<int> modeB{'u', 'k', 'v', 'h'};
    int nmodeA = modeA.size();
    int nmodeB = modeB.size();
    int nmodeC = modeC.size();

    // Extents
    // std::unordered_map<int, int64_t> extent;
    // extent['m'] = 96;
    // extent['n'] = 96;
    // extent['u'] = 96;
    // extent['v'] = 64;
    // extent['h'] = 64;
    // extent['k'] = 64;

    // Create a vector of extents for each tensor
    std::vector<int64_t> extentC;
    for (auto mode : modeC)
        extentC.push_back(extent[mode]);
    std::vector<int64_t> extentA;
    for (auto mode : modeA)
        extentA.push_back(extent[mode]);
    std::vector<int64_t> extentB;
    for (auto mode : modeB)
        extentB.push_back(extent[mode]);

    printf("Define modes and extents\n");

    /* ***************************** */

    // Number of elements of each tensor
    size_t elementsA = 1;
    for (auto mode : modeA)
        elementsA *= extent[mode];
    size_t elementsB = 1;
    for (auto mode : modeB)
        elementsB *= extent[mode];
    size_t elementsC = 1;
    for (auto mode : modeC)
        elementsC *= extent[mode];

    // Size in bytes
    size_t sizeA = sizeof(floatTypeA) * elementsA;
    size_t sizeB = sizeof(floatTypeB) * elementsB;
    size_t sizeC = sizeof(floatTypeC) * elementsC;

    // Allocate on device
    void *A_d, *B_d, *C_d;
    hipMalloc((void **)&A_d, sizeA);
    hipMalloc((void **)&B_d, sizeB);
    hipMalloc((void **)&C_d, sizeC);

    // Allocate on host
    floatTypeA *A = (floatTypeA *)malloc(sizeof(floatTypeA) * elementsA);
    floatTypeB *B = (floatTypeB *)malloc(sizeof(floatTypeB) * elementsB);
    floatTypeC *C = (floatTypeC *)malloc(sizeof(floatTypeC) * elementsC);

    // Initialize data on host
    for (int64_t i = 0; i < elementsA; i++)
        A[i] = (((float)rand()) / RAND_MAX - 0.5) * 100;
    for (int64_t i = 0; i < elementsB; i++)
        B[i] = (((float)rand()) / RAND_MAX - 0.5) * 100;
    for (int64_t i = 0; i < elementsC; i++)
        C[i] = (((float)rand()) / RAND_MAX - 0.5) * 100;

    // Copy to device
    HANDLE_CUDA_ERROR(hipMemcpy(A_d, A, sizeA, hipMemcpyHostToDevice));
    HANDLE_CUDA_ERROR(hipMemcpy(B_d, B, sizeB, hipMemcpyHostToDevice));
    HANDLE_CUDA_ERROR(hipMemcpy(C_d, C, sizeC, hipMemcpyHostToDevice));

    const uint32_t kAlignment = 128; // Alignment of the global-memory device pointers (bytes)
    assert(uintptr_t(A_d) % kAlignment == 0);
    assert(uintptr_t(B_d) % kAlignment == 0);
    assert(uintptr_t(C_d) % kAlignment == 0);

    printf("Allocate, initialize and transfer tensors\n");

    /*************************
     * cuTENSOR
     *************************/

    hiptensorHandle_t handle;
    HANDLE_ERROR(hiptensorCreate(&handle));

    /**********************
     * Create Tensor Descriptors
     **********************/

    hiptensorTensorDescriptor_t descA;
    HANDLE_ERROR(hiptensorCreateTensorDescriptor(handle,
                                                &descA,
                                                nmodeA,
                                                extentA.data(),
                                                NULL, /*stride*/
                                                typeA, kAlignment));

    hiptensorTensorDescriptor_t descB;
    HANDLE_ERROR(hiptensorCreateTensorDescriptor(handle,
                                                &descB,
                                                nmodeB,
                                                extentB.data(),
                                                NULL, /*stride*/
                                                typeB, kAlignment));

    hiptensorTensorDescriptor_t descC;
    HANDLE_ERROR(hiptensorCreateTensorDescriptor(handle,
                                                &descC,
                                                nmodeC,
                                                extentC.data(),
                                                NULL, /*stride*/
                                                typeC, kAlignment));

    printf("Initialize cuTENSOR and tensor descriptors\n");

    /*******************************
     * Create Contraction Descriptor
     *******************************/

    hiptensorOperationDescriptor_t desc;
    HANDLE_ERROR(hiptensorCreateContraction(handle,
                                           &desc,
                                           descA, modeA.data(), /* unary operator A*/ HIPTENSOR_OP_IDENTITY,
                                           descB, modeB.data(), /* unary operator B*/ HIPTENSOR_OP_IDENTITY,
                                           descC, modeC.data(), /* unary operator C*/ HIPTENSOR_OP_IDENTITY,
                                           descC, modeC.data(),
                                           descCompute));

    /*****************************
     * Optional (but recommended): ensure that the scalar type is correct.
     *****************************/

    hiptensorDataType_t scalarType;
    HANDLE_ERROR(hiptensorOperationDescriptorGetAttribute(handle,
                                                         desc,
                                                         HIPTENSOR_OPERATION_DESCRIPTOR_SCALAR_TYPE,
                                                         (void *)&scalarType,
                                                         sizeof(scalarType)));

    assert(scalarType == HIPTENSOR_R_32F);
    typedef float floatTypeCompute;
    floatTypeCompute alpha = (floatTypeCompute)1.1f;
    floatTypeCompute beta = (floatTypeCompute)0.f;

    /**************************
     * Set the algorithm to use
     ***************************/

    const hiptensorAlgo_t algo = HIPTENSOR_ALGO_DEFAULT;

    hiptensorPlanPreference_t planPref;
    HANDLE_ERROR(hiptensorCreatePlanPreference(
        handle,
        &planPref,
        algo,
        HIPTENSOR_JIT_MODE_NONE));

    /**********************
     * Query workspace estimate
     **********************/

    uint64_t workspaceSizeEstimate = 0;
    const hiptensorWorksizePreference_t workspacePref = HIPTENSOR_WORKSPACE_DEFAULT;
    HANDLE_ERROR(hiptensorEstimateWorkspaceSize(handle,
                                               desc,
                                               planPref,
                                               workspacePref,
                                               &workspaceSizeEstimate));

    /**************************
     * Create Contraction Plan
     **************************/

    hiptensorPlan_t plan;
    HANDLE_ERROR(hiptensorCreatePlan(handle,
                                    &plan,
                                    desc,
                                    planPref,
                                    workspaceSizeEstimate));

    /**************************
     * Optional: Query information about the created plan
     **************************/

    // query actually used workspace
    uint64_t actualWorkspaceSize = 0;
    HANDLE_ERROR(hiptensorPlanGetAttribute(handle,
                                          plan,
                                          HIPTENSOR_PLAN_REQUIRED_WORKSPACE,
                                          &actualWorkspaceSize,
                                          sizeof(actualWorkspaceSize)));

    // At this point the user knows exactly how much memory is need by the operation and
    // only the smaller actual workspace needs to be allocated
    assert(actualWorkspaceSize <= workspaceSizeEstimate);

    void *work = nullptr;
    if (actualWorkspaceSize > 0)
    {
        HANDLE_CUDA_ERROR(hipMalloc(&work, actualWorkspaceSize));
        assert(uintptr_t(work) % 128 == 0); // workspace must be aligned to 128 byte-boundary
    }

    /**********************
     * Execute
     **********************/

    hipStream_t stream;
    HANDLE_CUDA_ERROR(hipStreamCreate(&stream));

    HANDLE_ERROR(hiptensorContract(handle,
                                  plan,
                                  (void *)&alpha, A_d, B_d,
                                  (void *)&beta, C_d, C_d,
                                  work, actualWorkspaceSize, stream));

    /**********************
     * Free allocated data
     **********************/
    HANDLE_ERROR(hiptensorDestroy(handle));
    HANDLE_ERROR(hiptensorDestroyPlan(plan));
    HANDLE_ERROR(hiptensorDestroyOperationDescriptor(desc));
    HANDLE_ERROR(hiptensorDestroyTensorDescriptor(descA));
    HANDLE_ERROR(hiptensorDestroyTensorDescriptor(descB));
    HANDLE_ERROR(hiptensorDestroyTensorDescriptor(descC));
    HANDLE_CUDA_ERROR(hipStreamDestroy(stream));

    if (A)
        free(A);
    if (B)
        free(B);
    if (C)
        free(C);
    if (A_d)
        hipFree(A_d);
    if (B_d)
        hipFree(B_d);
    if (C_d)
        hipFree(C_d);
    if (work)
        hipFree(work);
}

void run(std::vector<char> modeC, std::vector<char> modeA, std::vector<char> modeB, std::unordered_map<char, int64_t> extent)
{
    // Convert char vectors to int vectors
    std::vector<int> modeC_int(modeC.begin(), modeC.end());
    std::vector<int> modeA_int(modeA.begin(), modeA.end());
    std::vector<int> modeB_int(modeB.begin(), modeB.end());

    // convert extent to std::unordered_map<int, int64_t>
    std::unordered_map<int, int64_t> extent_int;
    for (auto const &x : extent)
        extent_int[x.first] = x.second;

    performContraction(modeC_int, modeA_int, modeB_int, extent_int);
    HANDLE_CUDA_ERROR(hipDeviceSynchronize());
}