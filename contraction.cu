#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>
#include <assert.h>

#include <hip/hip_runtime.h>
#include <hiptensor.h>

#include <unordered_map>
#include <vector>
#include "contraction.cuh"

// Handle cuTENSOR errors
#define HANDLE_ERROR(x)                                         \
    {                                                           \
        const auto err = x;                                     \
        if (err != HIPTENSOR_STATUS_SUCCESS)                     \
        {                                                       \
            printf("Error: %s\n", hiptensorGetErrorString(err)); \
            exit(-1);                                           \
        }                                                       \
    };

#define HANDLE_CUDA_ERROR(x)                                \
    {                                                       \
        const auto err = x;                                 \
        if (err != hipSuccess)                             \
        {                                                   \
            printf("Error: %s\n", hipGetErrorString(err)); \
            exit(-1);                                       \
        }                                                   \
    };

std::vector<double> performContraction(std::vector<int> modeC, std::vector<int> modeA, std::vector<int> modeB, std::unordered_map<int, int64_t> extent, hiptensorAlgo_t algo, hiptensorDataType_t dataType = HIPTENSOR_R_16F)
{
    // Host element type definition
    typedef float floatTypeCompute;
    hiptensorDataType_t typeA, typeB, typeC;
    size_t elementSize;

    hiptensorComputeDescriptor_t descCompute;

    if (dataType == HIPTENSOR_R_16F)
    {
        printf("Running with FP16\n");
        typeA = HIPTENSOR_R_16F;
        typeB = HIPTENSOR_R_16F;
        typeC = HIPTENSOR_R_16F;
        elementSize = sizeof(_Float16);

        descCompute = HIPTENSOR_COMPUTE_DESC_16F;
    }
    else
    {
        typeA = HIPTENSOR_R_32F;
        typeB = HIPTENSOR_R_32F;
        typeC = HIPTENSOR_R_32F;
        elementSize = sizeof(float);

        descCompute = HIPTENSOR_COMPUTE_DESC_32F;
    }

    int nmodeA = modeA.size();
    int nmodeB = modeB.size();
    int nmodeC = modeC.size();

    std::vector<int64_t> extentC;
    for (auto mode : modeC)
        extentC.push_back(extent[mode]);
    std::vector<int64_t> extentA;
    for (auto mode : modeA)
        extentA.push_back(extent[mode]);
    std::vector<int64_t> extentB;
    for (auto mode : modeB)
        extentB.push_back(extent[mode]);

    double gflops = 1;
    for (auto const &x : extent)
    {
        gflops *= x.second;
    }
    gflops = gflops / 1e9;

    size_t elementsA = 1;
    for (auto mode : modeA)
        elementsA *= extent[mode];
    size_t elementsB = 1;
    for (auto mode : modeB)
        elementsB *= extent[mode];
    size_t elementsC = 1;
    for (auto mode : modeC)
        elementsC *= extent[mode];

    size_t sizeA = elementSize * elementsA;
    size_t sizeB = elementSize * elementsB;
    size_t sizeC = elementSize * elementsC;

    void *A_d, *B_d, *C_d;
    hipMalloc((void **)&A_d, sizeA);
    hipMalloc((void **)&B_d, sizeB);
    hipMalloc((void **)&C_d, sizeC);

    void *A = malloc(elementSize * elementsA);
    void *B = malloc(elementSize * elementsB);
    void *C = malloc(elementSize * elementsC);

    // if (dataType == HIPTENSOR_R_16F)
    // {
    //     hiprand_uniform(NULL, (__half *)A_d, elementsA);
    //     hiprand_uniform(NULL, (__half *)B_d, elementsB);
    //     hiprand_uniform(NULL, (__half *)C_d, elementsC);
    // }
    // else
    // {
    //     hiprand_uniform(NULL, (float *)A_d, elementsA);
    //     hiprand_uniform(NULL, (float *)B_d, elementsB);
    //     hiprand_uniform(NULL, (float *)C_d, elementsC);
    // }

    if (dataType == HIPTENSOR_R_16F)
    {
        for (int64_t i = 0; i < elementsA; i++)
            ((_Float16 *)A)[i] = ((_Float16)rand()) / RAND_MAX;
        for (int64_t i = 0; i < elementsB; i++)
            ((_Float16 *)B)[i] = ((_Float16)rand()) / RAND_MAX;
        for (int64_t i = 0; i < elementsC; i++)
            ((_Float16 *)C)[i] = ((_Float16)rand()) / RAND_MAX;
    }
    else
    {
        for (int64_t i = 0; i < elementsA; i++)
            ((float *)A)[i] = (((float)rand()) / RAND_MAX - 0.5) * 100;
        for (int64_t i = 0; i < elementsB; i++)
            ((float *)B)[i] = (((float)rand()) / RAND_MAX - 0.5) * 100;
        for (int64_t i = 0; i < elementsC; i++)
            ((float *)C)[i] = (((float)rand()) / RAND_MAX - 0.5) * 100;
    }

    HANDLE_CUDA_ERROR(hipMemcpy(A_d, A, sizeA, hipMemcpyHostToDevice));
    HANDLE_CUDA_ERROR(hipMemcpy(B_d, B, sizeB, hipMemcpyHostToDevice));
    HANDLE_CUDA_ERROR(hipMemcpy(C_d, C, sizeC, hipMemcpyHostToDevice));

    const uint32_t kAlignment = 128;
    assert(uintptr_t(A_d) % kAlignment == 0);
    assert(uintptr_t(B_d) % kAlignment == 0);
    assert(uintptr_t(C_d) % kAlignment == 0);

    hiptensorHandle_t handle;
    HANDLE_ERROR(hiptensorCreate(&handle));

    hiptensorTensorDescriptor_t descA;
    HANDLE_ERROR(hiptensorCreateTensorDescriptor(handle,
                                                &descA,
                                                nmodeA,
                                                extentA.data(),
                                                NULL,
                                                typeA, kAlignment));

    hiptensorTensorDescriptor_t descB;
    HANDLE_ERROR(hiptensorCreateTensorDescriptor(handle,
                                                &descB,
                                                nmodeB,
                                                extentB.data(),
                                                NULL,
                                                typeB, kAlignment));

    hiptensorTensorDescriptor_t descC;
    HANDLE_ERROR(hiptensorCreateTensorDescriptor(handle,
                                                &descC,
                                                nmodeC,
                                                extentC.data(),
                                                NULL,
                                                typeC, kAlignment));

    hiptensorOperationDescriptor_t desc;
    HANDLE_ERROR(hiptensorCreateContraction(handle,
                                           &desc,
                                           descA, modeA.data(), /* unary operator A*/ HIPTENSOR_OP_IDENTITY,
                                           descB, modeB.data(), /* unary operator B*/ HIPTENSOR_OP_IDENTITY,
                                           descC, modeC.data(), /* unary operator C*/ HIPTENSOR_OP_IDENTITY,
                                           descC, modeC.data(),
                                           descCompute));

    HANDLE_ERROR(hiptensorOperationDescriptorSetAttribute(handle,
                                                         &desc,
                                                         HIPTENSOR_OPERATION_DESCRIPTOR_SCALAR_TYPE,
                                                         (void *)&dataType,
                                                         sizeof(dataType)));

    hiptensorDataType_t scalarType;
    HANDLE_ERROR(hiptensorOperationDescriptorGetAttribute(handle,
                                                         desc,
                                                         HIPTENSOR_OPERATION_DESCRIPTOR_SCALAR_TYPE,
                                                         (void *)&scalarType,
                                                         sizeof(scalarType)));

    printf("Scalar type: %d\n", scalarType);
    printf("Data type: %d\n", dataType);

    assert(scalarType == dataType);
    floatTypeCompute alpha = (floatTypeCompute)1.0f;
    floatTypeCompute beta = (floatTypeCompute)0.f;

    hiptensorPlanPreference_t planPref;
    HANDLE_ERROR(hiptensorCreatePlanPreference(
        handle,
        &planPref,
        algo,
        HIPTENSOR_JIT_MODE_NONE));

    uint64_t workspaceSizeEstimate = 0;
    const hiptensorWorksizePreference_t workspacePref = HIPTENSOR_WORKSPACE_DEFAULT;
    HANDLE_ERROR(hiptensorEstimateWorkspaceSize(handle,
                                               desc,
                                               planPref,
                                               workspacePref,
                                               &workspaceSizeEstimate));

    hiptensorPlan_t plan;
    HANDLE_ERROR(hiptensorCreatePlan(handle,
                                    &plan,
                                    desc,
                                    planPref,
                                    workspaceSizeEstimate));

    uint64_t actualWorkspaceSize = 0;
    HANDLE_ERROR(hiptensorPlanGetAttribute(handle,
                                          plan,
                                          HIPTENSOR_PLAN_REQUIRED_WORKSPACE,
                                          &actualWorkspaceSize,
                                          sizeof(actualWorkspaceSize)));

    assert(actualWorkspaceSize <= workspaceSizeEstimate);

    void *work = nullptr;
    if (actualWorkspaceSize > 0)
    {
        HANDLE_CUDA_ERROR(hipMalloc(&work, actualWorkspaceSize));
        assert(uintptr_t(work) % 128 == 0);
    }

    hipStream_t stream;
    HANDLE_CUDA_ERROR(hipStreamCreate(&stream));

    float timing = 0;

    for (int i = 0; i < 5; i++)
    {
        HANDLE_CUDA_ERROR(hipMemcpy(C_d, C, sizeC, hipMemcpyHostToDevice));

        hipEvent_t start, stop;
        HANDLE_CUDA_ERROR(hipEventCreate(&start));
        HANDLE_CUDA_ERROR(hipEventCreate(&stop));
        HANDLE_CUDA_ERROR(hipEventRecord(start, stream));

        HANDLE_ERROR(hiptensorContract(handle,
                                      plan,
                                      (void *)&alpha, A_d, B_d,
                                      (void *)&beta, C_d, C_d,
                                      work, actualWorkspaceSize, stream));

        HANDLE_CUDA_ERROR(hipEventRecord(stop, stream));
        HANDLE_CUDA_ERROR(hipEventSynchronize(stop));
        float milliseconds = 0;
        HANDLE_CUDA_ERROR(hipEventElapsedTime(&milliseconds, start, stop));
        milliseconds /= 1000;
        timing += milliseconds;
    }

    timing /= 5;

    HANDLE_ERROR(hiptensorDestroy(handle));
    HANDLE_ERROR(hiptensorDestroyPlan(plan));
    HANDLE_ERROR(hiptensorDestroyOperationDescriptor(desc));
    HANDLE_ERROR(hiptensorDestroyTensorDescriptor(descA));
    HANDLE_ERROR(hiptensorDestroyTensorDescriptor(descB));
    HANDLE_ERROR(hiptensorDestroyTensorDescriptor(descC));
    HANDLE_CUDA_ERROR(hipStreamDestroy(stream));

    if (A)
        free(A);
    if (B)
        free(B);
    if (C)
        free(C);
    if (A_d)
        hipFree(A_d);
    if (B_d)
        hipFree(B_d);
    if (C_d)
        hipFree(C_d);
    if (work)
        hipFree(work);

    printf("Timing: %f s\n", timing);
    printf("GFLOPS: %f\n", gflops / timing);

    std::vector<double> returnVector = {(double)timing, gflops / timing};

    return returnVector;
}

std::vector<std::vector<double>> run(std::vector<char> modeC, std::vector<char> modeA, std::vector<char> modeB, std::unordered_map<char, int64_t> extent, hiptensorDataType_t dataType = HIPTENSOR_R_16F)
{
    std::vector<int> modeC_int(modeC.begin(), modeC.end());
    std::vector<int> modeA_int(modeA.begin(), modeA.end());
    std::vector<int> modeB_int(modeB.begin(), modeB.end());

    std::unordered_map<int, int64_t> extent_int;
    for (auto const &x : extent)
        extent_int[x.first] = x.second;

    hiptensorAlgo_t algo = HIPTENSOR_ALGO_DEFAULT;

    std::vector<double> returnVec1, returnVec2, returnVec3, returnVec4, returnVec5;

    returnVec1 = performContraction(modeC_int, modeA_int, modeB_int, extent_int, algo, dataType);
    HANDLE_CUDA_ERROR(hipDeviceSynchronize());

    algo = CUTENSOR_ALGO_GETT;

    returnVec2 = performContraction(modeC_int, modeA_int, modeB_int, extent_int, algo, dataType);
    HANDLE_CUDA_ERROR(hipDeviceSynchronize());

    algo = CUTENSOR_ALGO_TGETT;

    returnVec3 = performContraction(modeC_int, modeA_int, modeB_int, extent_int, algo, dataType);
    HANDLE_CUDA_ERROR(hipDeviceSynchronize());

    algo = CUTENSOR_ALGO_TTGT;

    returnVec4 = performContraction(modeC_int, modeA_int, modeB_int, extent_int, algo, dataType);
    HANDLE_CUDA_ERROR(hipDeviceSynchronize());

    algo = HIPTENSOR_ALGO_DEFAULT_PATIENT;

    returnVec5 = performContraction(modeC_int, modeA_int, modeB_int, extent_int, algo, dataType);
    HANDLE_CUDA_ERROR(hipDeviceSynchronize());

    std::vector<std::vector<double>> returnVec = {returnVec1, returnVec2, returnVec3, returnVec4, returnVec5};
    return returnVec;
}